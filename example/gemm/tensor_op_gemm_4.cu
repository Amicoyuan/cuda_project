#include <iostream>
#include <hip/hip_runtime.h>
#include "mma.h"
#include "hip/hip_fp16.h"

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16
#define WARP_SIZE 32
#define WARP_X 4
#define WARP_Y 2
#define M_TILE 128
#define K_TILE 32
#define N_TILE 256
#define THREAD_PER_BLOCK 256

// 向量化访存，预取 4 * 4B ,  8 * 2B, 即 8个half
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

using namespace nvcuda;

// 在这里我们一个block 分了32个线程
// 一个warp 刚好调度32个线程
// tensor op 的抽象层级刚刚好是 warp level
// 而 cuda core op的抽象则是 thread level

// tensor core 相关的op都在  nvcuda:wmma  命名空间下
// 详情见：https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma-description 

/*
A100 SM80 FP16的最高算力是 312 TFLOPS
一共有108个SM 
根据计算能力为 8.0 的 A100 GPU, 每个 SM 最多可以支持 64 个并发 warps, 而对于计算能力为 8.6 的 GPU，每个 SM 可以支持最多 48 个并发 warps
一个SM有4个Tensor core
*/

__global__ void sgemm_warp_tensor_op(const int M, const int N, const int K,
    void * __restrict__ d_A, void * __restrict__ d_B, void * __restrict__ d_C
    ) {
    

    half * a = reinterpret_cast<half *>((void *)d_A);
    half * b = reinterpret_cast<half *>((void *)d_B);
    half * c = reinterpret_cast<half *>((void *)d_C);

    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;

    // 右移 5位 相当于 tid / 32
    int wid = tid >> 5;  // warp_tid  当前有8个warp

    // 每个 warp 算的 C_tile 为 [M_64, N_64]

    const int APAD = 8;  // pad 避免 bank conflict
    const int BPAD = 8;
    

    // NV 共有 32个 bank , 一个 bank 4B。 总共 128 B
    // float4 是 4 * 4 = 16B，
    // 但是 FLOAT4 是8个half,  
    // 这里就需要 pad 4 * 4B = 16B, 刚好对应者 2 * 8个half = 16B 
    __shared__ half s_a[BM][BK + APAD];
    __shared__ half s_b[BK][BN + BPAD];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_b[2][4];
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> frag_c[4][4];

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }
    /*
    tid = 0-3   load_a_smem_m 0
    tid = 4-7   load_a_smem_m 2
    tid = 124-127 load load_a_smem_m 62
    tid = 252-255 load load_a_smem_m 126
    */
    int load_a_smem_m = (tid >> 2) << 1;    // 除 4 * 2
    /*
    // 每个线程对应拿8个half
    tid     load_a_smem_k
    0       0
    1       8
    2      16
    3      24
    */
    int load_a_smem_k = (tid &  3) << 3;    // 与操作 再 * 8 = tid 的最后两位乘以 8

    /*
    tid     load_b_smem_k
    0       0
    1       0
    ...
    31      0
    32      4
    ...
    63      4
    64      8
    ...
    95      8
    96      12
    ...
    127     12
    128     16
    ...
    159     16
    160     20
    ...
    191     20
    192     24
    ...
    223     24
    224     28
    ...
    255     28
    */
    int load_b_smem_k = (tid >> 5) << 2; // 除以32 * 4
    /*
    tid     load_b_smem_n
    0       0
    1       8
    2      16
    3      24
    ...
    27     216
    28     224
    29     232
    30     240
    31     248
    32      0
    33     16
    ...
    61     248
    62      0
    63     248
    64      0
    ...
    255   248
    */
    int load_b_smem_n = (tid & 31) << 3; // 这个表达式相当于将 tid 的最低5位乘以 8

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    for (int bk = 0; bk < K / BK; bk++) {
        // 每 4 个线程拿 A [M_2, K_32]
        FLOAT4(s_a[load_a_smem_m    ][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr        ]);
        FLOAT4(s_a[load_a_smem_m + 1][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr +     K]);

        // 每 32 个线程  拿 B [ K_4, N_256 ]
        FLOAT4(s_b[load_b_smem_k    ][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr        ]);
        FLOAT4(s_b[load_b_smem_k + 1][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr +     N]);
        FLOAT4(s_b[load_b_smem_k + 2][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr + 2 * N]);
        FLOAT4(s_b[load_b_smem_k + 3][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr + 3 * N]);
        


        // 偏移指正 下轮 K迭代
        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        __syncthreads();
        

        // warp tensor op 读 smem
        wmma::load_matrix_sync(frag_a[0][0], &s_a[comp_c_frag_m * 64     ][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[comp_c_frag_m * 64 + 16][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[comp_c_frag_m * 64 + 32][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[comp_c_frag_m * 64 + 48][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[comp_c_frag_m * 64     ][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[comp_c_frag_m * 64 + 16][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[comp_c_frag_m * 64 + 32][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[comp_c_frag_m * 64 + 48][16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[ 0][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[ 0][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[ 0][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[ 0][comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[16][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[16][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[16][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[16][comp_c_frag_n * 64 + 48], BN + BPAD);



        // A行B列  增加数据复用
        #pragma unroll
        for (int i = 0; i < 4; i++) {
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }

        __syncthreads();
    }


    // 写回C
    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}


int main() {
    int M = 4096; // Example value
    int N = 8192; // Example value
    int K = 8192; // Example value

    // Allocate host memory
    float *h_A_ft32 = new float[M * K];
    float *h_B_ft32 = new float[K * N];
    float *h_C_ft32 = new float[M * N];

    uint16_t *h_A_ft16 = new uint16_t[M * K];
    uint16_t *h_B_ft16 = new uint16_t[K * N];
    uint16_t *h_C_ft16 = new uint16_t[M * N];

    printf("init data....\n");



    printf("Host compute....\n"); 


    printf("Device compute....\n"); 
    
    // Allocate device memory
    void *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(uint16_t));
    hipMalloc(&d_B, K * N * sizeof(uint16_t));
    hipMalloc(&d_C, M * N * sizeof(uint16_t));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A_ft16, M * K * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_ft16, K * N * sizeof(uint16_t), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 block(THREAD_PER_BLOCK);


    std::cout << "M_TILE  N_TILE " << M_TILE <<" "<< N_TILE<<std::endl;
    
    dim3 grid( ((N + N_TILE - 1) / N_TILE), ((M + M_TILE - 1) / M_TILE));

    std::cout << "blockDimx  blockDimy " << ((N + N_TILE - 1) / N_TILE) <<" "<< ((M + M_TILE - 1) / M_TILE)<<std::endl;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Launch kernel
    sgemm_warp_tensor_op<<<grid, block>>>(M, N, K, d_A, d_B, d_C);

    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Time taken by kernel: " << elapsedTime << " milliseconds" << std::endl;
    std::cout << "M N K " << M <<" "<< N<<" "<< K<<std::endl;
    // Copy result back to host
    hipMemcpy(h_C_ft16, d_C, M * N * sizeof(uint16_t), hipMemcpyDeviceToHost);

    // Compute FLOPS
    float flops = 2.0f * M * N * K;
    float timeInSeconds = elapsedTime / 1000.0f; // Convert milliseconds to seconds
    float gflops = (flops / 1e9f) / timeInSeconds; // FLOPS to GFLOPS

    std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;

    // Cleanup
    delete[] h_A_ft32;
    delete[] h_B_ft32;
    delete[] h_C_ft32;
    delete[] h_A_ft16;
    delete[] h_B_ft16;
    delete[] h_C_ft16;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}