#include <iostream>
#include <hip/hip_runtime.h>
#include "mma.h"
#include "hip/hip_fp16.h"

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16
#define WARP_SIZE 32
#define TENSOR_OP_M 2
#define TENSOR_OP_N 2
#define M_TILE 32
#define N_TILE 32

using namespace nvcuda;

// 在这里我们一个block 分了32个线程
// 一个warp 刚好调度32个线程
// tensor op 的抽象层级刚刚好是 warp level
// 而 cuda core op的抽象则是 thread level

// tensor core 相关的op都在  nvcuda:wmma  命名空间下
// 详情见：https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma-description 

/*
A100 SM80 FP16的最高算力是 312 TFLOPS
一共有108个SM 
根据计算能力为 8.0 的 A100 GPU, 每个 SM 最多可以支持 64 个并发 warps, 而对于计算能力为 8.6 的 GPU，每个 SM 可以支持最多 48 个并发 warps
一个SM有4个Tensor core
*/

__global__ void sgemm_warp_tensor_op(int M, int N, int K, void *__restrict__ d_A, void *__restrict__ d_B, void *__restrict__ d_C) {
    
    
    half * A = reinterpret_cast<half *>((void *)d_A);
    half * B = reinterpret_cast<half *>((void *)d_B);
    half * C = reinterpret_cast<half *>((void *)d_C);
    
    const int K_tiles = (K + WMMA_K - 1) / WMMA_K;


    // 而我们现在这个例子,  一个block有 128个线程, 共有 4个 warp  ,目的是利用上 一个sm中的4个tensor core
    // 每个block算的C_tile 变为 32 * 32
    // 一个warp负责 16 * 16的 C_tile_frag
    const int warp_id = threadIdx.x / WARP_SIZE;
    const int warp_x_id = warp_id % TENSOR_OP_N;
    const int warp_y_id = warp_id / TENSOR_OP_N;
    const int warp_row = blockIdx.y * M_TILE;
    const int warp_col = blockIdx.x * N_TILE;

    if (warp_row >= M || warp_col >= N) {
        return;
    }

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> C_frag;
    wmma::fill_fragment(C_frag, 0.0f);




#pragma unroll
    for (size_t i = 0; i < K_tiles; i++) {
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> A_frag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> B_frag;
        // ldm描述连续行（对于行主要布局）或列（对于列主要布局）之间的元素跨度
        // 在这里即为K
        
        // 这里偏移很好理解了
        wmma::load_matrix_sync(A_frag, A + warp_row * K + i * WMMA_K + warp_y_id * K * WMMA_M, K); 
        wmma::load_matrix_sync(B_frag, B + i * WMMA_K + warp_col * K + warp_x_id * K * WMMA_N, K);

        wmma::mma_sync(C_frag, A_frag, B_frag, C_frag);
    }
    wmma::store_matrix_sync(C + warp_y_id * WMMA_M * N + warp_row * N + warp_x_id * WMMA_N + warp_col  , C_frag, N, wmma::mem_row_major);
}

int main() {
    int M = 2048; // Example value
    int N = 2048; // Example value
    int K = 2048; // Example value

    // Allocate host memory
    float *h_A_ft32 = new float[M * K];
    float *h_B_ft32 = new float[K * N];
    float *h_C_ft32 = new float[M * N];

    uint16_t *h_A_ft16 = new uint16_t[M * K];
    uint16_t *h_B_ft16 = new uint16_t[K * N];
    uint16_t *h_C_ft16 = new uint16_t[M * N];

    printf("init data....\n");



    printf("Host compute....\n"); 


    printf("Device compute....\n"); 
    
    // Allocate device memory
    void *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(uint16_t));
    hipMalloc(&d_B, K * N * sizeof(uint16_t));
    hipMalloc(&d_C, M * N * sizeof(uint16_t));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A_ft16, M * K * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_ft16, K * N * sizeof(uint16_t), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 block(WARP_SIZE * TENSOR_OP_M * TENSOR_OP_N);


    std::cout << "M_TILE  N_TILE " << M_TILE <<" "<< N_TILE<<std::endl;
    
    dim3 grid( ((N + N_TILE - 1) / N_TILE), ((M + M_TILE - 1) / M_TILE));

    std::cout << "blockDimx  blockDimy " << ((N + N_TILE - 1) / N_TILE) <<" "<< ((M + M_TILE - 1) / M_TILE)<<std::endl;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Launch kernel
    sgemm_warp_tensor_op<<<grid, block>>>(M, N, K, d_A, d_B, d_C);

    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Time taken by kernel: " << elapsedTime << " milliseconds" << std::endl;
    std::cout << "M N K " << M <<" "<< N<<" "<< K<<std::endl;
    // Copy result back to host
    hipMemcpy(h_C_ft16, d_C, M * N * sizeof(uint16_t), hipMemcpyDeviceToHost);

    // Compute FLOPS
    float flops = 2.0f * M * N * K;
    float timeInSeconds = elapsedTime / 1000.0f; // Convert milliseconds to seconds
    float gflops = (flops / 1e9f) / timeInSeconds; // FLOPS to GFLOPS

    std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;

    // Cleanup
    delete[] h_A_ft32;
    delete[] h_B_ft32;
    delete[] h_C_ft32;
    delete[] h_A_ft16;
    delete[] h_B_ft16;
    delete[] h_C_ft16;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}