#include <iostream>
#include <hip/hip_runtime.h>

__global__ void sgemm(int M, int N, int K, float *A, float *B, float *C) {
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    if (m < M && n < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; i++) {
            sum += A[m * K + i] * B[i * N + n];
        }
        C[m * N + n] = sum;
    }
}

int main() {
    int M = 1024; // Example value
    int N = 1024; // Example value
    int K = 1024; // Example value

    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];

    // Initialize host matrices
    // (Omitted for brevity; you'd typically fill these matrices with data here)

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 block(16, 16);
    dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Launch kernel
    sgemm<<<grid, block>>>(M, N, K, d_A, d_B, d_C);

    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Time taken by kernel: " << elapsedTime << " milliseconds" << std::endl;

    // Copy result back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Compute FLOPS
    float flops = 2.0f * M * N * K;
    float timeInSeconds = elapsedTime / 1000.0f; // Convert milliseconds to seconds
    float gflops = (flops / 1e9f) / timeInSeconds; // FLOPS to GFLOPS

    std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;

    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
